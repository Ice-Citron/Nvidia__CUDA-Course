
#include <hip/hip_runtime.h>
#include <stdio.h>

__global__ void printSuccessForCorrectExecutionConfiguration()
{

  if(threadIdx.x == 1023 && blockIdx.x == 255)
  {
    printf("Success!\n");
  } 
  else {
    // printf("Failure. Update the execution configuration as necessary.\n");
  }
}

int main()
{
  /*
   * Update the execution configuration so that the kernel
   * will print `"Success!"`.
   */

  printSuccessForCorrectExecutionConfiguration<<<256, 1024>>>();
  // Don't forget that the kernel execution is asynchronous, and you must sync on its completion
  
  hipDeviceSynchronize();
}
